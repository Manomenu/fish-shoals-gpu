#include "hip/hip_runtime.h"
#include "fish_model.cuh"

FishModel::FishModel()
{
	float sqrt3 = glm::sqrt(3);
	std::vector<float> vertices =
	{
		h, 0, 0,
		0, 0, a * sqrt3 / 3,
		0, -a / 2, -a * sqrt3 / 6,
		0,	a / 2, -a * sqrt3 / 6,
	};
	std::copy(vertices.begin(), vertices.end(), this->vertices);

	std::vector<int> indices = {
		0, 1, 2,
		0, 2, 3,
		0, 1, 3,
		1, 2, 3
	};
	std::copy(indices.begin(), indices.end(), this->indices);
}